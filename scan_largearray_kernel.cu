#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>
//#include <scan_largearray.cu>


#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.
#define BLOCK_SIZE 256
#define DEFAULT_NUM_ELEMENTS 512
#define STEPS DEFAULT_NUM_ELEMENTS/BLOCK_SIZE

// Lab4: Host Helper Functions (allocate your own data structure...)
float* AllocateDeviceArray(float * A)
{
	float * Adevice = A;
	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	hipMalloc((void**)&Adevice, size);
	return Adevice;
}



void CopyToDeviceArray(float * Adevice, float * Ahost)
{
	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	hipMemcpy(Adevice, Ahost, size,hipMemcpyHostToDevice);
}

void CopyFromDeviceArray(float * Ahost, float * Adevice)
{
	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	hipMemcpy(Ahost, Adevice, size, hipMemcpyDeviceToHost);
}

// Lab4: Device Functions


// Lab4: Kernel Functions
__global__ void computeKernel( float* odata, float* idata, unsigned int len)
{

	__shared__ float temp[BLOCK_SIZE]; // allocated on invocation
	__device__ float sums[STEPS];
	__device int my_block_count = 0;
	temp[0] = 0;
	int stride = 1;

	__shared__ unsigned int my_blockId;
	if (threadIdx.x==0)
	{
		my_blockId = atomicInc( &my_block_count, (unsigned int) -1 );
	}

	int tid = __mul24(threadIdx.y, 16)+threadIdx.x;
	int bid = blockIdx.x;
	int i_element = __mul24(bid,BLOCK_SIZE)+tid;

	double total_sum = 0;

	for (int j = 1; j < BLOCK_SIZE; j++)
	{
		total_sum += idata[j];
		temp[j] = temp[j-1]+idata[j-1];
	}
	syncthreads();
	sums[my_blockId] = total_sum;

	int partial_sum = 0;

	for(int i = 0; i < my_blockId; i++)
		partial_sum += sums[i];
	for(int j = 0; j < BLOCK_SIZE; j++)
		odata[j+(my_blockId*BLOCK_SIZE)] = partial_sum + temp[j];
	syncthreads();
	//if (total_sum != odata[len-1])
	//printf("Warning: exceeding single-precision accuracy.  Scan will be inaccurate.\n");
}



// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	float * answer;
	// float * Adevice_in = AllocateDeviceArray(inArray);
	// CopyToDeviceArray(Adevice_in, inArray);
	// float * Adevice_out = AllocateDeviceArray(outArray);
	// CopyToDeviceArray(Adevice_out, outArray);

	dim3 dimGrid(DEFAULT_NUM_ELEMENTS/BLOCK_SIZE,1);
	dim3 dimBlock(16,16);

	unsigned int len = DEFAULT_NUM_ELEMENTS;
	computeKernel <<< dimGrid, dimBlock >>> (outArray , inArray, len);
	//hipDeviceSynchronize();
	//CopyFromDeviceArray(answer, outArray);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
