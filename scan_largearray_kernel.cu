#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>
//#include <scan_largearray.cu>


#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.
#define BLOCK_SIZE 256
#define DEFAULT_NUM_ELEMENTS 16000000

// Lab4: Host Helper Functions (allocate your own data structure...)
float* AllocateDeviceArray(float * A)
{
	float * Adevice = A;
	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	hipMalloc((void**)&Adevice, size);
	return Adevice;
}

// Allocate a matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.
//	If init == 1, perform random initialization.

// Copy a host matrix to a device matrix.
void CopyToDeviceArray(float * Adevice, float * Ahost)
{
	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	hipMemcpy(Adevice, Ahost, size,hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void CopyFromDeviceArray(float * Ahost, float * Adevice)
{
	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	hipMemcpy(Ahost, Adevice, size, hipMemcpyDeviceToHost);
}

// Lab4: Device Functions


// Lab4: Kernel Functions
__global__ void computeKernel( float* reference, float* idata, unsigned int len)
{
	// reference[0] = 0;
	// double total_sum = 0;
	// for( unsigned int i = 1; i < len; ++i)
	// {
	//     total_sum += idata[i-1];
	//     reference[i] = idata[i-1] + reference[i-1];
	// }
	// if (total_sum != reference[len-1])
	//     printf("Warning: exceeding single-precision accuracy.  Scan will be inaccurate.\n");

}



// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	float * Adevice_in = AllocateDeviceArray(inArray);
	CopyToDeviceArray(Adevice_in, inArray);
	float * Adevice_out = AllocateDeviceArray(outArray);
	CopyToDeviceArray(Adevice_out, outArray);

	dim3 dimGrid(DEFAULT_NUM_ELEMENTS/BLOCK_SIZE,1);
	dim3 dimBlock(16,16);

 	unsigned int len = DEFAULT_NUM_ELEMENTS;
	computeKernel << dimGrid, dimBlock >>> (Adevice_out ,Adevice_in, len);
	hipDeviceSynchronize();
	CopyFromDeviceArray(outArray, Adevice_out);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
