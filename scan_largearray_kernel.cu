#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>
#include "CONSTANTS.h"


<<<<<<< Updated upstream
=======

>>>>>>> Stashed changes
// Lab4: Host Helper Functions (allocate your own data structure...)

// Lab4: Device Functions


// Lab4: Kernel Functions
__global__ void computeKernel( float* odata, float* idata, unsigned int len)
{
	
	__shared__ int bid;
	
	int partial_sum = 0;
	bid  = blockIdx.x;
	__shared__ float temp[DEFAULT_NUM_ELEMENTS];
	temp[0] = 0;
	odata[0] = 0;
 	double total_sum;
	
	
	unsigned int tid = __mul24(threadIdx.y, 16) + threadIdx.x;
	unsigned int element;
	
	  	for(int j = 1; j < DEFAULT_NUM_ELEMENTS; j++)
	  	{ 		
	  		//element = __mul24(BLOCK_SIZE, blockIdx.x)+ j;
	  		total_sum += idata[j];
			temp[j] = temp[j-1]+idata[j-1];
			
	  	}
	  	
	  	syncthreads();

	  	
	  	
	  	for(int j = 0; j < DEFAULT_NUM_ELEMENTS; j++)
	  	{
	  	//element = __mul24(BLOCK_SIZE, blockIdx.x)+ j;
	  	odata[j] = temp[j];
	  	      
	  	}
	  	syncthreads();
	  
}



// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	dim3 dimGrid(DEFAULT_NUM_ELEMENTS/BLOCK_SIZE,1);
	dim3 dimBlock(16,16);

	unsigned int len = DEFAULT_NUM_ELEMENTS;
	computeKernel <<< dimGrid, dimBlock >>> (outArray , inArray, len);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_

