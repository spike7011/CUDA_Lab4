#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>
//#include <scan_largearray.cu>


#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.
#define BLOCK_SIZE 256
#define DEFAULT_NUM_ELEMENTS 512
#define STEPS DEFAULT_NUM_ELEMENTS/BLOCK_SIZE

// Lab4: Host Helper Functions (allocate your own data structure...)

// Lab4: Device Functions

__device__ float sums[STEPS];
__device__ unsigned int count = 0;
// Lab4: Kernel Functions
__global__ void computeKernel( float* odata, float* idata, unsigned int len)
{
	
	__shared__ int bid;
	
	int partial_sum = 0;
	bid  = blockIdx.x;
	__shared__ float temp[BLOCK_SIZE];
	temp[0] = 0;
	__shared__ double total_sum;
	
	
	unsigned int tid = __mul24(threadIdx.y, 16) + threadIdx.x;
	unsigned int element;
	
	  	for(int j = 0; j < BLOCK_SIZE; j++)
	  	{
	  		if (j== 0)
	  			total_sum = 0;
	  		else
	  		{
	  		element = __mul24(BLOCK_SIZE, blockIdx.x)+ j;
	  		total_sum += idata[element];
			temp[j] = temp[j-1]+idata[element-1];
			}
	  	}
	  	sums[bid] = total_sum;
	  	syncthreads();

	  	if(bid != 0)
	  	{
	  	partial_sum += sums[STEPS-bid];
	  	odata[bid*BLOCK_SIZE] = partial_sum;
	  	}
	  	else 
	  	{
	  	odata[bid] = 0;
	  	partial_sum = 0;
	  	}
	  	
	  	for(int j = 0; j < BLOCK_SIZE; j++)
	  	{
	  	 element = __mul24(BLOCK_SIZE, blockIdx.x)+ j;
	  	 	if(j == 0)
	  	 	odata[element] = partial_sum;
	  	      else
	  	       odata[element] = temp[j] + partial_sum;
	  	      
	  	}
	  	syncthreads();
	  
}



// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	//int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	//float * answer;
	// float * Adevice_in = AllocateDeviceArray(inArray);
	// CopyToDeviceArray(Adevice_in, inArray);
	// float * Adevice_out = AllocateDeviceArray(outArray);
	// CopyToDeviceArray(Adevice_out, outArray);

	dim3 dimGrid(DEFAULT_NUM_ELEMENTS/BLOCK_SIZE,1);
	dim3 dimBlock(16,16);

	unsigned int len = DEFAULT_NUM_ELEMENTS;
	computeKernel <<< dimGrid, dimBlock >>> (outArray , inArray, len);
	//hipDeviceSynchronize();
	//CopyFromDeviceArray(answer, outArray);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_

