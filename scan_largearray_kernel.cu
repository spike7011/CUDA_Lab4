#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>
#include <stdint.h>
#define BLOCK_SIZE  1024  //128 or 32 seem to be safe and work for all sizes

// Lab4: Host Helper Functions (allocate your own data structure...)

// Lab4: Device Functions
//__device__ uint32_t count = 0;    //keeps track of number of launched blocks
__device__ uint32_t finished_blocks = 0;  // keeps track of which blocks are finished doing local scan
__device__ uint32_t partial[100000];


__global__ void computeKernel( float* odata, float* idata, unsigned int len, uint32_t STEPS)
{
    uint32_t tid = threadIdx.x;
    uint32_t index;
    partial[0]=0;
    __shared__ uint32_t temp[BLOCK_SIZE];
    index = __mul24(BLOCK_SIZE, blockIdx.x);
    temp[tid]= *(idata+index+tid);
    syncthreads();

    uint32_t stride = 1;

    while (stride < BLOCK_SIZE)
    {
        int index = __mul24(tid+1,stride*2) - 1;
        if (index < BLOCK_SIZE)
            temp[index] = temp[index] + temp[index-stride];
        stride = stride << 1;
        syncthreads();
    }

    syncthreads();
    stride = BLOCK_SIZE;
    while(stride > 1)
    {
        int index = __mul24(tid+1,stride) - 1;
        stride = stride >> 1;
        if(index+stride < BLOCK_SIZE)
            temp[index+stride] += temp[index];
        syncthreads();
    }

    while(finished_blocks < blockIdx.x)
        syncthreads();

	float pa=0;

    if(tid == 0)
    {

        if (blockIdx.x<STEPS-1)
            partial[blockIdx.x+1] = partial[blockIdx.x]+temp[BLOCK_SIZE-1];

        atomicInc(&finished_blocks, STEPS);
    }
    syncthreads();

    if ( blockIdx.x>0 )
        pa=partial[blockIdx.x];

    if (index+tid+1 < len)
        odata[index+tid+1] =  temp[tid] + pa;
    else
        odata[0] = 0;
}

// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{
  uint32_t STEPS = ((numElements-1)/BLOCK_SIZE)+1;

	dim3 dimGrid(STEPS,1);
	dim3 dimBlock(BLOCK_SIZE,1);


	computeKernel <<< dimGrid, dimBlock >>> (outArray , inArray, numElements, STEPS);
}
// **===-----------------------------------------------------------===**




#endif // _PRESCAN_CU_
