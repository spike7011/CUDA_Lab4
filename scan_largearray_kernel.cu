#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>


#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.
#define BLOCK_SIZE 256

// Lab4: Host Helper Functions (allocate your own data structure...)


// Lab4: Device Functions


// Lab4: Kernel Functions


// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	extern __shared__ float temp[];// allocated on invocation
	int thid = threadIdx.x;
	int offset = 1;
	temp[2*thid] = inArray[2*thid]; // load input into shared memory
	temp[2*thid+1] = inArray[2*thid+1];
	for (int d = numElements>>1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	if (thid == 0) { temp[numElements - 1] = 0; } // clear the last element
	for (int d = 1; d < numElements; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	outArray[2*thid] = temp[2*thid]; // write results to device memory
	outArray[2*thid+1] = temp[2*thid+1];

}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
