#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>
#include "CONSTANTS.h"


// Lab4: Host Helper Functions (allocate your own data structure...)

// Lab4: Device Functions
__device__ unsigned int count = 0;
__device__ float partial_sum = 0;

// Lab4: Kernel Functions
__global__ void computeKernel( float* odata, float* idata, unsigned int len)
{

	__shared__ unsigned int mbid;
	__shared__ float temp[BLOCK_SIZE];
	float partial[STEPS];
	
	unsigned int tid = threadIdx.x+__mul24(16,threadIdx.y);
	if(tid == 0)
	{
	mbid = atomicInc(&count, (unsigned int) -1);
	}
	syncthreads();
	
	unsigned int bid = blockIdx.x;
	
	odata[0] = 0;
 	__shared__ double block_sum;
 	
	unsigned int element;
	
	
	
	for(int i = 0; i < STEPS; i++)
	{
			block_sum = 0;
			temp[0] = 0;
		 	for(int j = 1; j < BLOCK_SIZE; j++)
		  	{ 		
		  		element = __mul24(BLOCK_SIZE, bid)+ j;
		  		block_sum += idata[element];
				temp[j] = temp[j-1]+idata[element-1];
			
		  	}
		  	if(i== 0)
		  	partial[i] = 0;
		  	else partial[i] = block_sum+partial[i-1];
		  	syncthreads();
	 }
		
	for(int i = 0; i < STEPS;i++)
	if(i == mbid-1)
	{
	  	for(int j = 0; j < BLOCK_SIZE; j++)
	  	{
	  	element = __mul24(BLOCK_SIZE, i)+ j;
	  	odata[element] = temp[j]+partial[i];
	  	}
	  	syncthreads();
	  }
} 
	 




// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	dim3 dimGrid(DEFAULT_NUM_ELEMENTS/BLOCK_SIZE,1);
	dim3 dimBlock(16,16);

	unsigned int len = DEFAULT_NUM_ELEMENTS;
	computeKernel <<< dimGrid, dimBlock >>> (outArray , inArray, len);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_

