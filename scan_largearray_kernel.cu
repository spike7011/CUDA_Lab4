#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>
#include "CONSTANTS.h"


// Lab4: Host Helper Functions (allocate your own data structure...)

// Lab4: Device Functions
__device__ int count = -1;
__device__ float partial[STEPS];
//__device__ double global_block_sum = 0;

// Lab4: Kernel Functions
__global__ void computeKernel( float* odata, float* idata, unsigned int len)
{
	int tid = __mul24(threadIdx.y, 16) + threadIdx.x;
	int bid = blockIdx.x;
	int element;
	
	__shared__  int mbid;
	if(tid == 0)
		mbid = atomicAdd(&count, 1);
	syncthreads();
		
	__shared__ float temp[BLOCK_SIZE];
	
	
	odata[0] = 0;
 	
 	
	
	temp[0] = 0;
	
	syncthreads();
	
	//each thread block does a partial summation
	for(int i = 0; i < STEPS; i++)
	{
		 	for(int j = 1; j < BLOCK_SIZE; j++)
		  	{ 		
		  		element = __mul24(BLOCK_SIZE, bid)+ j;
				temp[j] = temp[j-1]+idata[element-1];
			
		  	}
		  	syncthreads();
	 }
	 //end of parallel sums per TB
	 
		
	
	for(int i = 0; i < STEPS; i++)
		if (i == 0)
		{
			for(int j = 0; j < BLOCK_SIZE; j++)
		  	{
		  	odata[j] = temp[j];
		  	//odata[j] = bid;
		  	}
		  	partial[0] = temp[BLOCK_SIZE-1]+idata[BLOCK_SIZE-1];
		  	syncthreads();
		  	
		}
		else
		{
		        //partial[0] += temp[0];
		  	for(int j = 0; j < BLOCK_SIZE; j++)
		  	{
		  	element = __mul24(BLOCK_SIZE, 1)+ j;
		  	odata[element] = partial[0]+temp[j];
		  	//odata[element]= bid;
		  	}
		  	partial[1] = temp[BLOCK_SIZE-1];
		  	syncthreads();
		  	
		}	
		syncthreads();
	}
	




// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	dim3 dimGrid(DEFAULT_NUM_ELEMENTS/BLOCK_SIZE,1);
	dim3 dimBlock(16,16);

	unsigned int len = DEFAULT_NUM_ELEMENTS;
	computeKernel <<< dimGrid, dimBlock >>> (outArray , inArray, len);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_

