#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>
#include "CONSTANTS.h"


// Lab4: Host Helper Functions (allocate your own data structure...)

// Lab4: Device Functions
__device__ int count = -1;    //keeps track of number of launched blocks
__device__ int count2 = -1;  // keeps track of which blocks are finished doing local scan
__device__ float partial[STEPS];
//__device__ double global_block_sum = 0;

// Lab4: Kernel Functions
__global__ void computeKernel( float* odata, float* idata, unsigned int len)
{
	int tid = __mul24(threadIdx.y, 16) + threadIdx.x;
	int bid = blockIdx.x;
	if(bid == 0 && tid == 0)
		odata[0] = 0;
	__shared__  int mbid;
	if(tid == 0)
		mbid = atomicAdd(&count, 1);
	syncthreads();
	//each thread block obtains it's local blockId in the shared variable mbid
	
	
	
	int element;
	
	
		
	__shared__ float temp[BLOCK_SIZE];
	
	//each thread block does a partial summation
	for(int i = 0; i < STEPS; i++)
	{
			temp[0] = 0;
		 	for(int j = 1; j < BLOCK_SIZE; j++)
		  	{ 		
		  		element = __mul24(BLOCK_SIZE, bid)+ j;
				temp[j] = temp[j-1]+idata[element-1];
				
		  	}
		  	
		  	
		  	syncthreads();
		  	
	 }
	 //syncthreads();
	 //end of parall+el sums per TB
	 
	__shared__  int mbid_done;
	if(tid == 0)
		mbid_done = atomicAdd(&count2, 1);
	syncthreads();
	
	
	
	for(int i = 0; i < STEPS; i++)
		if (bid == 0 )
		{
			for(int j = 0; j < BLOCK_SIZE; j++)
		  	{
		  	odata[j] = temp[j];
		  	//odata[j] = bid;
		  	}
		  	partial[0] = temp[BLOCK_SIZE-1]+idata[BLOCK_SIZE-1];
		  	//syncthreads();
		  	
		}
		else if (bid <= count2)
		{
		        //partial[0] += temp[0];
		  	for(int j = 0; j < BLOCK_SIZE; j++)
		  	{
		  	element = __mul24(BLOCK_SIZE, bid)+ j;
		  	odata[element] = temp[j] + partial[bid-1];
		  	//odata[element]= partial[bid-1];
		  	}
		  	partial[bid] = temp[BLOCK_SIZE-1]+partial[bid-1];//+idata[BLOCK_SIZE*i-1];
		  	//syncthreads();
		  	
		}	
		syncthreads();
	}
	




// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	dim3 dimGrid(DEFAULT_NUM_ELEMENTS/BLOCK_SIZE,1);
	dim3 dimBlock(16,16);

	unsigned int len = DEFAULT_NUM_ELEMENTS;
	computeKernel <<< dimGrid, dimBlock >>> (outArray , inArray, len);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_

