#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>
//#include <scan_largearray.cu>


#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.
#define BLOCK_SIZE 256
#define DEFAULT_NUM_ELEMENTS 256

// Lab4: Host Helper Functions (allocate your own data structure...)
float* AllocateDeviceArray(float * A)
{
	float * Adevice = A;
	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	hipMalloc((void**)&Adevice, size);
	return Adevice;
}



void CopyToDeviceArray(float * Adevice, float * Ahost)
{
	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	hipMemcpy(Adevice, Ahost, size,hipMemcpyHostToDevice);
}

void CopyFromDeviceArray(float * Ahost, float * Adevice)
{
	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	hipMemcpy(Ahost, Adevice, size, hipMemcpyDeviceToHost);
}

// Lab4: Device Functions


// Lab4: Kernel Functions
__global__ void computeKernel( float* odata, float* idata, unsigned int len)
{

	__shared__ float temp[BLOCK_SIZE]; // allocated on invocation
	temp[0] = 0;
	int stride = 1;
	int tid = __mul24(threadIdx.y, 16)+threadIdx.x;
	int bid = blockIdx.x;
	int i_element = __mul24(bid,BLOCK_SIZE)+tid;

	odata[0] = 0;
	double total_sum = 0;

	if(tid==0)
	{
		for(unsigned int i = 1; i < len; ++i)
		{
			total_sum += idata[i-1];
			odata[i] = idata[i-1] + odata[i-1];
		}
	}

		syncthreads();
		//if (total_sum != odata[len-1])
		//printf("Warning: exceeding single-precision accuracy.  Scan will be inaccurate.\n");
}



// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	int size = DEFAULT_NUM_ELEMENTS * sizeof(float);
	float * answer;
	// float * Adevice_in = AllocateDeviceArray(inArray);
	// CopyToDeviceArray(Adevice_in, inArray);
	// float * Adevice_out = AllocateDeviceArray(outArray);
	// CopyToDeviceArray(Adevice_out, outArray);

	dim3 dimGrid(DEFAULT_NUM_ELEMENTS/BLOCK_SIZE,1);
	dim3 dimBlock(16,16);

	unsigned int len = DEFAULT_NUM_ELEMENTS;
	computeKernel <<< dimGrid, dimBlock >>> (outArray , inArray, len);
	//hipDeviceSynchronize();
	//CopyFromDeviceArray(answer, outArray);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
