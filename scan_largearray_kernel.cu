#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>



// Lab4: Host Helper Functions (allocate your own data structure...)

// Lab4: Device Functions
__device__ uint32_t count = 0;    //keeps track of number of launched blocks
__device__ uint32_t count2 = 0;  // keeps track of which blocks are finished doing local scan
__device__ float partial[100];
//__device__ double global_block_sum = 0;

// Lab4: Kernel Functions
__global__ void computeKernel( float* odata, float* idata, unsigned int len)
{
	uint32_t tid = threadIdx.x;
	uint32_t bid = blockIdx.x;
	__shared__ uint32_t index;
	
	__shared__ uint32_t  mbid;
	__shared__ float temp[BLOCK_SIZE];
	
	if(tid == 0)
	{
		mbid = atomicInc(&count, (unsigned int) -1);
		index = __mul24(BLOCK_SIZE, mbid);
		temp[0]=0;
	 	for(int j = 1; j < BLOCK_SIZE; j++)
  	{ 		
			temp[j] = temp[j-1]+idata[index + j - 1];
	  }
		partial[mbid] = temp[BLOCK_SIZE-1] + idata[index + BLOCK_SIZE-1];
	  
	  //index = __mul24(BLOCK_SIZE, mbid);
		}
   
   if(tid == 0)
     atomicInc(&count2, (unsigned int) -1 );
   syncthreads();
   int done = 0;
  
	
	float p = 0;
	if (mbid>0) 
    for (int o=0;o<mbid;o++)
      p += partial[o];
	odata[index+tid] = p + temp[tid] ;
	
  syncthreads();
  
 
}
	
__global__ void computeKernel_o1( float* odata, float* idata, unsigned int len)
{

	
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(bid == 0 && tid == 0)
		odata[0] = 0;
	__shared__  int mbid;
	if(tid == 0)
		mbid = atomicAdd(&count, 1);
	syncthreads();
	//each thread block obtains it's local blockId in the shared variable mbid
	
	
	
	int element;
	
	
		
	__shared__ float temp[BLOCK_SIZE+1];
	for (int j = 0; j < STEPS; j++)
	{
		int stride = 2;
		
		memcpy(temp, idata+j*BLOCK_SIZE, sizeof(float)*BLOCK_SIZE);
		while(stride < PRINT_NUM)
		{
		if(tid<BLOCK_SIZE)
			if((tid+1)%stride == 0)
				temp[tid] = temp[tid] + temp[tid-stride/2];
		syncthreads();
		stride*=2;
		}
		
		//post scan step
		stride /=2 ;
		while(stride > 1)
		{
		if(tid < BLOCK_SIZE && tid != 0)
			if(tid - stride >= 0)
					if((tid-stride)%(stride/2) == 0)
						temp[tid] += temp[tid-stride/2];
		syncthreads();
		stride /= 2;
		
		}
		
			
	 }//syncthreads();
	
	 
	
	
	
	
	for(int i = 0; i < STEPS; i++)
	{
		if (bid == 0 )
		{
			for(int j = 0; j < BLOCK_SIZE; j++)
		  	{
		  	odata[j] = temp[j];
		  	//odata[j] = bid;
		  	}
		  	partial[0] = temp[BLOCK_SIZE-1]+idata[BLOCK_SIZE-1];
		  	//syncthreads();
		  	
		}
		/*else if (bid <= count2)
		{
		        //partial[0] += temp[0];
		  	for(int j = 0; j < 8; j++)
		  	{
		  	element = __mul24(BLOCK_SIZE, bid)+ j;
		  	odata[element] = temp[j]; //+ partial[bid-1];
		  	//odata[element]= partial[bid-1];
		  	}
		  	partial[bid] = temp[BLOCK_SIZE-1]+partial[bid-1];//+idata[BLOCK_SIZE*i-1];
		  	//syncthreads();
		  	
		}	*/
		syncthreads();
		
	}
	
}
	




// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	dim3 dimGrid(STEPS,1);
	dim3 dimBlock(BLOCK_SIZE,1);

	unsigned int len = DEFAULT_NUM_ELEMENTS;
	computeKernel <<< dimGrid, dimBlock >>> (outArray , inArray, len);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_

