#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>



// Lab4: Host Helper Functions (allocate your own data structure...)

// Lab4: Device Functions
__device__ uint32_t count = 0;    //keeps track of number of launched blocks
__device__ uint32_t count2 = 0;  // keeps track of which blocks are finished doing local scan
__device__ uint32_t done = 0;
__device__ double partial[STEPS];

// Lab4: Kernel Functions
__global__ void computeKernel( float* odata, float* idata, unsigned int len)
{
	uint32_t tid = threadIdx.x;
	uint32_t bid = blockIdx.x;
	__shared__ uint32_t index;
	
	__shared__ uint32_t  mbid;
	__shared__ uint32_t  mbid2;
	__shared__ double temp[BLOCK_SIZE];
	__shared__ int prec;
	if(tid == 0)
	{
		mbid = atomicInc(&count, (unsigned int) -1);
		index = __mul24(BLOCK_SIZE, mbid);
		temp[0]=0;
	 	for(int j = 1; j < BLOCK_SIZE; j++)
  	{ 		
			temp[j] = temp[j-1]+idata[index + j - 1];
	  }
		partial[mbid] = temp[BLOCK_SIZE-1] + idata[index + BLOCK_SIZE-1];
	}
 
  while(count2 < mbid)
    syncthreads();

  mbid2= atomicInc(&count2, (unsigned int) -1 );   
 	
	__shared__ double p;
 	
  if(tid == 0)
	{
    p=0;
  	if (mbid>0) 
      for (int o=0;o<mbid;o++)
        p += partial[o];
  }

  syncthreads();
  
	odata[index+tid] =  p + temp[tid] ;
	
  syncthreads();
}
	


__global__ void computeKernel_o2( float* odata, float* idata, unsigned int len)
{
	uint32_t tid = threadIdx.x;
	uint32_t bid = blockIdx.x;
	__shared__ uint32_t index;
	
	__shared__ uint32_t  mbid;
	__shared__ uint32_t  mbid2;
	__shared__ float temp[BLOCK_SIZE];
	__shared__ int prec;
	if(tid == 0)
	{
   if(mbid == 0)
    odata[0]=0;
		mbid = atomicInc(&count, (unsigned int) -1);
  }
  syncthreads();
  
// magic begins
	index = __mul24(BLOCK_SIZE, mbid);
	temp[0]=0;
 
	//reduction step
	memcpy(temp, idata+index, sizeof(float)*BLOCK_SIZE);
  int stride = 1;
  while (stride < BLOCK_SIZE)
  {
     int pos = (tid+1)*stride*2 -1;
     if (pos < BLOCK_SIZE)
       if((pos-stride) >= 0)
       temp[pos] = temp[pos] + temp[pos-stride];
     stride = stride*2;
    syncthreads();
  }
  
  
  
  stride = BLOCK_SIZE / 2;
  while(stride > 0)
  {
    int index = (threadIdx.x+1)*stride*2 - 1;
    if(index < BLOCK_SIZE) 
      temp[index+stride] = temp[index] + temp[index+stride];
    stride /= 2;
  syncthreads();
  }
   
  partial[mbid] = temp[BLOCK_SIZE-1];
 
 	/*or(int j = 1; j < BLOCK_SIZE; j++)
 	{ 		
			      
      //start modifiying here
      
      temp[j] = temp[j-1]+idata[index + j - 1];
  }
	partial[mbid] = temp[BLOCK_SIZE-1] + idata[index + BLOCK_SIZE-1];
*/

// magic ends
 
  while(count2 < mbid)
    syncthreads();

  mbid2= atomicInc(&count2, (unsigned int) -1 );   
 	
	__shared__ double p;
 	
  if(tid == 0)
	{
    p=0;
  	if (mbid>0) 
      for (int o=0;o<mbid;o++)
        p += partial[o];
  }

  syncthreads();
  
	odata[index+tid+1] =  temp[tid] + p;
	
  syncthreads();
}
	




// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	dim3 dimGrid(STEPS,1);
	dim3 dimBlock(BLOCK_SIZE,1);

	unsigned int len = DEFAULT_NUM_ELEMENTS;
	computeKernel <<< dimGrid, dimBlock >>> (outArray , inArray, len);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_

